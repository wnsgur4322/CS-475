#include "hip/hip_runtime.h"
/*
CS 475 - Project #5
CUDA Monte Carlo
author: Junhyeok Jeong
email: jeongju@oregonstate.edu
*/

// System includes
#include <stdio.h>
#include <assert.h>
#include <malloc.h>
#include <math.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>

using namespace std;

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"


// setting the number of trials in the monte carlo simulation:
#ifndef NUMTRIALS
#define NUMTRIALS	( 1024*1024 )
//#define NUMTRIALS	131072
#endif


#ifndef BLOCKSIZE
#define BLOCKSIZE		32     // number of threads per block
#endif

#define NUMBLOCKS		( NUMTRIALS / BLOCKSIZE )


// ranges for the random numbers:
// give about 42% probability

const float XCMIN =	 0.0;
const float XCMAX =	 2.0;
const float YCMIN =	 0.0;
const float YCMAX =	 2.0;
const float RMIN  =	 0.5;
const float RMAX  =	 2.0;

// range from project #1
// ranges for the random numbers:
// give about 13% probability
/*
const float XCMIN =	-1.0;
const float XCMAX =	 1.0;
const float YCMIN =	 0.0;
const float YCMAX =	 2.0;
const float RMIN  =	 0.5;
const float RMAX  =	 2.0;
*/

// function prototypes:
float		Ranf( float, float );
int		Ranf( int, int );
void		TimeOfDaySeed( );



__global__  void MonteCarlo( float *Xcs, float *Ycs, float *Rs, int *Hits )
{
	unsigned int wgNumber      = blockIdx.x;
	unsigned int wgDimension   = blockDim.x;
	unsigned int threadNum     = threadIdx.x;
	unsigned int gid           = wgNumber*wgDimension + threadNum;

	// all the monte carlo stuff goes in here
	// if we make it all the way through, then Hits[gid] = 1

	// randomize the location and radius of the circle:
	float xc = Xcs[gid];
	float yc = Ycs[gid];
	float  r =  Rs[gid];

	float tn = tanf( (float)( (M_PI/180.) * 30. ) );
	Hits[gid] = 0;

	// solve for the intersection using the quadratic formula:

	float a = 1. + tn*tn;
	float b = -2.*( xc + yc*tn );
	float c = xc*xc + yc*yc - r*r;
	float d = b*b - 4.*a*c;

	// cascading if-statements:
	//	if you used "continue;" in project #1, change to this style because,
	//	if there is no for-loop, then there is nowhere to continue to

	// If d is less than 0., then the circle was completely missed. (Case A) Continue on to the next trial in the for-loop.
	if( d >= 0. ){
		// hits the circle:
		// get the first intersection:
		d = sqrt( d );
		float t1 = (-b + d ) / ( 2.*a );	// time to intersect the circle
		float t2 = (-b - d ) / ( 2.*a );	// time to intersect the circle
		float tmin = t1 < t2 ? t1 : t2;		// only care about the first intersection


		if( tmin >= 0. ){
		// where does it intersect the circle?
		    float xcir = tmin;
		    float ycir = tmin*tn;

		// get the unitized normal vector at the point of intersection:
		    float nx = xcir - xc;
		    float ny = ycir - yc;
		    float n = sqrt( nx*nx + ny*ny );
		    nx /= n;	// unit vector
		    ny /= n;	// unit vector

		// get the unitized incoming vector:
		    float inx = xcir - 0.;
		    float iny = ycir - 0.;
		    float in = sqrt( inx*inx + iny*iny );
		    inx /= in;	// unit vector
		    iny /= in;	// unit vector

		// get the outgoing (bounced) vector:
		    float dot = inx*nx + iny*ny;
		    float outx = inx - 2.*nx*dot;	// angle of reflection = angle of incidence`
		    float outy = iny - 2.*ny*dot;	// angle of reflection = angle of incidence`

		// find out if it hits the infinite plate:
			float t = ( 0. - ycir ) / outy;

			if( t >= 0. )
			{
				Hits[gid] = 1;
			}
		}
	}
}


// main program:

int
main( int argc, char* argv[ ] )
{
	TimeOfDaySeed( );

	int dev = findCudaDevice(argc, (const char **)argv);

	// allocate host memory:

	float *hXcs  = new float[NUMTRIALS];
	float *hYcs  = new float[NUMTRIALS];
	float * hRs  = new float[NUMTRIALS];
	int   *hHits = new   int[NUMTRIALS];

	// fill the random-value arrays:
	for( int n = 0; n < NUMTRIALS; n++ )
	{
		hXcs[n] = Ranf( XCMIN, XCMAX );
		hYcs[n] = Ranf( YCMIN, YCMAX );
 		hRs[n]  = Ranf(  RMIN,  RMAX );
	}

	// allocate device memory:

	float *dXcs, *dYcs, *dRs;
	int *dHits;

	dim3 dimsXcs(  NUMTRIALS, 1, 1 );
	dim3 dimsYcs(  NUMTRIALS, 1, 1 );
	dim3 dimsRs(   NUMTRIALS, 1, 1 );
	dim3 dimsHits( NUMTRIALS, 1, 1 );


	hipError_t status;
	status = hipMalloc( (void **)(&dXcs), NUMTRIALS*sizeof(float) );
	checkCudaErrors( status );

	status = hipMalloc( (void **)(&dYcs), NUMTRIALS*sizeof(float) );
	checkCudaErrors( status );

	status = hipMalloc( (void **)(&dRs), NUMTRIALS*sizeof(float) );
	checkCudaErrors( status );

	status = hipMalloc( (void **)(&dHits), NUMTRIALS *sizeof(int) );
	checkCudaErrors( status );


	// copy host memory to the device:

	status = hipMemcpy( dXcs, hXcs, NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
	checkCudaErrors( status );

	status = hipMemcpy( dYcs, hYcs, NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
	checkCudaErrors( status );

	status = hipMemcpy( dRs, hRs, NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
	checkCudaErrors( status );


	// setup the execution parameters:

	dim3 threads(BLOCKSIZE, 1, 1 );
	dim3 grid(NUMBLOCKS, 1, 1 );

	// create and start timer

	hipDeviceSynchronize( );

	// allocate CUDA events that we'll use for timing:

	hipEvent_t start, stop;
	status = hipEventCreate( &start );
	checkCudaErrors( status );
	status = hipEventCreate( &stop );
	checkCudaErrors( status );

	// record the start event:

	status = hipEventRecord( start, NULL );
	checkCudaErrors( status );

	// execute the kernel:

	MonteCarlo<<< grid, threads >>>( dXcs, dYcs, dRs, dHits );

	// record the stop event:

	status = hipEventRecord( stop, NULL );
	checkCudaErrors( status );

	// wait for the stop event to complete:

	status = hipEventSynchronize( stop );
	checkCudaErrors( status );

	float msecTotal = 0.0f;
	status = hipEventElapsedTime( &msecTotal, start, stop );
	checkCudaErrors( status );

	// compute and print the performance

	double secondsTotal = 0.001 * (double)msecTotal;
	double trialsPerSecond = (float)NUMTRIALS / secondsTotal;
	double megaTrialsPerSecond = trialsPerSecond / 1000000.;
	fprintf( stderr, "Number of Trials = %10d, MegaTrials/Second = %10.4lf\n", NUMTRIALS, megaTrialsPerSecond );

	// copy result from the device to the host:

	status = hipMemcpy( hHits, dHits, NUMTRIALS *sizeof(int), hipMemcpyDeviceToHost );
	checkCudaErrors( status );
	hipDeviceSynchronize( );

	// compute the probability:

	int numHits = 0;
	for(int i = 0; i < NUMTRIALS; i++ )
	{
		numHits += hHits[i];
	}

	float probability = 100.f * (float)numHits / (float)NUMTRIALS;
	fprintf(stderr, "\nProbability = %6.3f %%\n", probability );

	// clean up memory:
	delete [ ] hXcs;
	delete [ ] hYcs;
	delete [ ] hRs;
	delete [ ] hHits;

	status = hipFree( dXcs );
	status = hipFree( dYcs );
	status = hipFree( dRs );
	status = hipFree( dHits );
	checkCudaErrors( status );

	//write a result record file
    ofstream result;
    result.open("result.txt", ios::app);
    result << BLOCKSIZE << "\t" << NUMTRIALS << "\t" << probability << "\t" << megaTrialsPerSecond << "\t" << endl;
    result.close();

	return 0;
}

float
Ranf( float low, float high )
{
	float r = (float) rand();               // 0 - RAND_MAX
	float t = r  /  (float) RAND_MAX;       // 0. - 1.

	return   low  +  t * ( high - low );
}

int
Ranf( int ilow, int ihigh )
{
	float low = (float)ilow;
	float high = ceil( (float)ihigh );

	return (int) Ranf(low,high);
}

void
TimeOfDaySeed( )
{
	struct tm y2k = { 0 };
	y2k.tm_hour = 0;   y2k.tm_min = 0; y2k.tm_sec = 0;
	y2k.tm_year = 100; y2k.tm_mon = 0; y2k.tm_mday = 1;

	time_t  timer;
	time( &timer );
	double seconds = difftime( timer, mktime(&y2k) );
	unsigned int seed = (unsigned int)( 1000.*seconds );    // milliseconds
	srand( seed );
}